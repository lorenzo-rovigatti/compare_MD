#include "CUDA_device_utils.h"

#include <cstdio>
#include <cstdlib>

int get_device_count() {
	int deviceCount = 0;
	if(hipGetDeviceCount(&deviceCount) != hipSuccess) {
		fprintf(stderr, "hipGetDeviceCount FAILED, CUDA Driver and Runtime CUDA Driver and Runtime version may be mismatched, exiting.\n");
		exit(-1);
	}

	return deviceCount;
}

void check_device_existance(int device) {
	if(device >= get_device_count()) {
		fprintf(stderr, "The selected device doesn't exist, exiting.\n");
		exit(-1);
	}
}

hipDeviceProp_t get_current_device_prop() {
	int curr_dev;
	hipGetDevice(&curr_dev);
	return get_device_prop(curr_dev);
}

hipDeviceProp_t get_device_prop(int device) {
	check_device_existance(device);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	return deviceProp;
}

hipError_t set_device(int device) {
	check_device_existance(device);
	hipDeviceReset();
	return hipSetDevice(device);
}

hipDeviceProp_t set_device_automatically() {
	int trydev = 0;
	int ndev = -1;
	hipGetDeviceCount(&ndev);
	fprintf(stderr, "The computer has %i devices\n", ndev);
	while(trydev < ndev) {
		fprintf(stderr, " - Trying device %i\n", trydev);
		hipDeviceProp_t tryprop = get_device_prop(trydev);
		fprintf(stderr, " -- Device %i has properties %i.%i\n", trydev, tryprop.major, tryprop.minor);
		// we don't support old devices
		if (tryprop.major < 2 && tryprop.minor <= 2) {
			fprintf(stderr, " -- Device properties are not good. Skipping it\n");
			trydev ++;
			continue;
		}
		set_device (trydev);
		int *dummyptr = NULL;
		hipError_t test = hipMalloc((void **)&dummyptr, (size_t)sizeof(int));
		if(test == hipSuccess) {
			fprintf(stderr, " -- Using device %i\n", trydev);
			hipFree(dummyptr);
			break;
		}
		else {
			fprintf(stderr, " -- Device %i not available ...\n", trydev);
		}
		trydev++;
	}

	if(trydev == ndev) {
		fprintf(stderr, "No suitable devices available\n");
		exit(1);
	}

	fprintf(stderr, " --- Running on device %i\n", trydev);
	set_device(trydev);
	return get_device_prop(trydev);
}
