/*
 * CUDA.cpp
 *
 *  Created on: 14 giu 2018
 *      Author: lorenzo
 */

#include "CUDA.h"
#include "CUDA_device_utils.h"
#include "CUDA_kernels.cuh"

#include <cstdlib>
#include <cstdio>

#define CHECK_CUDA_ERROR(msg) __check_CUDA_error(msg, __FILE__, __LINE__)
inline void __check_CUDA_error(const char *errorMessage, const char *file, const int line) {
	hipError_t err = hipGetLastError();

	if(hipSuccess != err) {
		fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n", file, line, errorMessage, (int) err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void CUDA_init(MD_system *syst) {
	hipDeviceProp_t props = set_device_automatically();
	syst->kernel_threads_per_block = 2 * props.warpSize;

	syst->kernel_blocks.x = syst->N / syst->kernel_threads_per_block + ((syst->N % syst->kernel_threads_per_block == 0) ? 0 : 1);
	if(syst->kernel_blocks.x == 0) syst->kernel_blocks.x = 1;
	syst->kernel_blocks.y = syst->kernel_blocks.z = 1;

	fprintf(stderr, "CUDA device: %s\n", props.name);
	fprintf(stderr, "CUDA threads per block: %d\n", syst->kernel_threads_per_block);
	fprintf(stderr, "CUDA blocks: (%d, %d, %d)\n", syst->kernel_blocks.x, syst->kernel_blocks.y, syst->kernel_blocks.z);

	syst->vector_size = sizeof(vector) * syst->N;
	hipMalloc(&syst->d_positions, syst->vector_size);
	hipMalloc(&syst->d_velocities, syst->vector_size);
	hipMalloc(&syst->d_forces, syst->vector_size);

	hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &syst->N, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(MD_box_side), &syst->box_side, sizeof(number));

	CPU_to_CUDA(syst);
}

void CUDA_clean(MD_system *syst) {
	hipFree(&syst->d_positions);
	hipFree(&syst->d_velocities);
	hipFree(&syst->d_forces);
}

void CUDA_to_CPU(MD_system *syst) {
	hipMemcpy(syst->positions, syst->d_positions, syst->vector_size, hipMemcpyDeviceToHost);
	hipMemcpy(syst->velocities, syst->d_velocities, syst->vector_size, hipMemcpyDeviceToHost);
	hipMemcpy(syst->forces, syst->d_forces, syst->vector_size, hipMemcpyDeviceToHost);
}

void CPU_to_CUDA(MD_system *syst) {
	hipMemcpy(syst->d_positions, syst->positions, syst->vector_size, hipMemcpyHostToDevice);
	hipMemcpy(syst->d_velocities, syst->velocities, syst->vector_size, hipMemcpyHostToDevice);
	hipMemcpy(syst->d_forces, syst->forces, syst->vector_size, hipMemcpyHostToDevice);
}

void CUDA_first_step(MD_system *syst) {
	first_step_kernel
		<<<syst->kernel_blocks, syst->kernel_threads_per_block>>>
		(syst->d_positions, syst->d_velocities, syst->d_forces);
		CHECK_CUDA_ERROR("first_step error");
}

void CUDA_force_calculation(MD_system *syst) {
	force_calculation_kernel
		<<<syst->kernel_blocks, syst->kernel_threads_per_block>>>
		(syst->d_positions, syst->d_forces);
		CHECK_CUDA_ERROR("force_calculation error");
}

void CUDA_thermalise(MD_system *syst) {

}

void CUDA_second_step(MD_system *syst) {
	second_step_kernel
		<<<syst->kernel_blocks, syst->kernel_threads_per_block>>>
		(syst->d_velocities, syst->d_forces);
		CHECK_CUDA_ERROR("second_step error");
}
